﻿
#include <hip/hip_runtime.h>
#include <iostream>
#include <hip/hip_runtime_api.h>

__device__ int d_result;
__global__ void kernel(const int n)
{
	d_result = n;
}

int main()
{
    const int n = rand();
    kernel<<<1, 1>>>(n);
    const hipError_t cuda_status = hipGetLastError();
    if (cuda_status != hipSuccess)
    {
        std::cout << "FAIL: " << hipGetErrorString(cuda_status) << std::endl;
        return 1;
    }

    std::cout << "PASS";
    return 0;
}
